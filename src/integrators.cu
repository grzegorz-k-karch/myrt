#include "hip/hip_runtime.h"
#include "scene.h"
#include "intergrators.h"

__global__ void renderScene(Scene* scene, Integrator* integrator)
{
  integrator->render(scene);
}

